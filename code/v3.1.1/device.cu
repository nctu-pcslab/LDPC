#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "circuit.h"
#include "device.h"
using namespace std;
//extern CIRCUIT Circuit;

void CIRCUIT::CUDA_MemoryAllocate(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    CudaData = new CUDA_DATA;
    CudaData->h_DecodedData = new int[n_Total];       //redundant, for CPU debug
    hipMalloc((void**)&CudaData->d_DecodedData, n_Total * sizeof(int));
    ////// BitNode on host //////
    CudaData->h_LcxSize = new int [n_Total];
    CudaData->h_LcxBegin = new int [n_Total];
#ifdef DOUBLE
    CudaData->h_Lcx = new double [TotalEdge];
    CudaData->h_Lint = new double [n_Total];
#else
    CudaData->h_Lcx = new float [TotalEdge];
    CudaData->h_Lint = new float [n_Total];
#endif
    CudaData->h_LxcPosition = new int [TotalEdge];
    ////// CheckNode on host //////
    CudaData->h_LxcSize = new int [k_Total];
#ifdef DOUBLE
    CudaData->h_Lxc = new double [TotalEdge];
#else
    CudaData->h_Lxc = new float [TotalEdge];
#endif
    CudaData->h_LxcBegin = new int [k_Total];
    CudaData->h_LcxPosition = new int [TotalEdge];
    ////// BitNode on device //////
    hipMalloc((void**)&CudaData->d_LcxSize, n_Total * sizeof(int));
    hipMalloc((void**)&CudaData->d_LcxBegin, n_Total * sizeof(int));
#ifdef DOUBLE
    hipMalloc((void**)&CudaData->d_Lcx, TotalEdge * sizeof(double));
    hipMalloc((void**)&CudaData->d_Lint, n_Total * sizeof(double));
#else
    hipMalloc((void**)&CudaData->d_Lcx, TotalEdge * sizeof(float));
    hipMalloc((void**)&CudaData->d_Lint, n_Total * sizeof(float));
#endif
    hipMalloc((void**)&CudaData->d_LxcPosition, TotalEdge * sizeof(int));
    ////// CheckNode on device //////
    hipMalloc((void**)&CudaData->d_LxcSize, k_Total * sizeof(int));
#ifdef DOUBLE
    hipMalloc((void**)&CudaData->d_Lxc, TotalEdge * sizeof(double));
#else
    hipMalloc((void**)&CudaData->d_Lxc, TotalEdge * sizeof(float));
#endif
    hipMalloc((void**)&CudaData->d_LxcBegin, k_Total * sizeof(int));
    hipMalloc((void**)&CudaData->d_LcxPosition, TotalEdge * sizeof(int));
#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_MemoryAllocate);
#endif
}

void CIRCUIT::CUDA_CreateDataArray(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    int Begin=0;
    ///// BitNode /////
    for(unsigned i=0; i<n_Total; i++){
        CudaData->h_LcxSize[i] = BitNode[i]->Lcx.size();
        CudaData->h_LcxBegin[i] = Begin;
        //BitNode[i]->h_LcxBegin = Begin;
        Begin=Begin+BitNode[i]->Lcx.size();
        //cout<< BitNode[i]->ID<<endl;
    }
    ///// CheckNode //////
    Begin=0;
    for(unsigned i=0; i<k_Total; i++){
        CudaData->h_LxcSize[i] = CheckNode[i]->Lxc.size();
        CudaData->h_LxcBegin[i] = Begin;
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++){
            CudaData->h_LcxPosition[Begin+j] = CudaData->h_LcxBegin[CheckNode[i]->BitNode_H[j]->ID] + CheckNode[i]->Lcx_position[j];
        }
        Begin=Begin+CheckNode[i]->Lxc.size();
    }
    ///// BitNode /////
    for(unsigned i=0; i<n_Total; i++){
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++)
            CudaData->h_LxcPosition[CudaData->h_LcxBegin[i]+j] = CudaData->h_LxcBegin[BitNode[i]->CheckNode_H[j]->ID] + BitNode[i]->Lxc_position[j]; 
    }
#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_CreateDataArray);
#endif
}

void CIRCUIT::CUDA_CreateDataArray2(){
#ifdef PROFILE
    Timer.TimerStart();
#endif

    for(unsigned i=0; i<n_Total; i++){
        CudaData->h_Lint[i] = BitNode[i]->Lint;
        /*for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){
            CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j] = BitNode[i]->Lcx[j];
        }*/
    }
    for(unsigned i=0; i<k_Total; i++)
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++)
            CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j] = CheckNode[i]->Lxc[j];

#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_CreateDataArray2);
#endif
}

void CIRCUIT::UpdateLcx_CPU(){
    for(int i=0; i<k_Total; i++){
        for(int j=0; j<CudaData->h_LxcSize[i]; j++){
            double sgn=1;
            double minLxc=1000;
            for(int jj=0; jj<CudaData->h_LxcSize[i]; jj++){
                if(j != jj){
                    if(CudaData->h_Lxc[CudaData->h_LxcBegin[i]+jj] > 0)
                        sgn = sgn*1;
                    else
                        sgn = sgn*(-1);
                    minLxc = min(minLxc, fabs(CudaData->h_Lxc[CudaData->h_LxcBegin[i]+jj]));
                }
            }
            CudaData->h_Lcx[CudaData->h_LcxPosition[CudaData->h_LxcBegin[i]+j]] = sgn * minLxc;
        }
    }
}

void CIRCUIT::UpdateLxc_CPU(){
    for(int i=0; i<n_Total; i++){
        double sumLcx=0;
        for(int j=0; j<CudaData->h_LcxSize[i]; j++){
            sumLcx = sumLcx + CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j];
        }
        for(int j=0; j<CudaData->h_LcxSize[i]; j++){
             CudaData->h_Lxc[CudaData->h_LxcPosition[CudaData->h_LcxBegin[i]+j]] = CudaData->h_Lint[i] + (sumLcx - CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j]);
        }
    }
}

void CIRCUIT::Calculate_Posterior_CPU(){
    for(int i=0; i<n_Total; i++){
        double sumLcx=0;
        for(int j=0; j<CudaData->h_LcxSize[i]; j++)
            sumLcx = sumLcx + CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j];
        if(CudaData->h_Lint[i] + sumLcx >= 0)
            CudaData->h_DecodedData[i] = 0;
        else
            CudaData->h_DecodedData[i] = 1;
    }
}

void CIRCUIT::MemoryCopy_H2D(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    ////// BitNode on device //////
    hipMemcpy(CudaData->d_LcxSize, CudaData->h_LcxSize, n_Total * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LcxBegin, CudaData->h_LcxBegin, n_Total * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LxcPosition, CudaData->h_LxcPosition, TotalEdge * sizeof(int), hipMemcpyHostToDevice);
    ////// CheckNode on device //////
    hipMemcpy(CudaData->d_LxcSize, CudaData->h_LxcSize, k_Total * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LxcBegin, CudaData->h_LxcBegin, k_Total * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LcxPosition, CudaData->h_LcxPosition, TotalEdge * sizeof(int), hipMemcpyHostToDevice);

#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_H2D);
#endif
}

void CIRCUIT::MemoryCopy_H2D2(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    ////// BitNode on device //////
#ifdef DOUBLE
    //hipMemcpy(CudaData->d_Lcx, CudaData->h_Lcx, TotalEdge * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_Lint, CudaData->h_Lint, n_Total * sizeof(double), hipMemcpyHostToDevice);
    ////// CheckNode on device //////
    hipMemcpy(CudaData->d_Lxc, CudaData->h_Lxc, TotalEdge * sizeof(double), hipMemcpyHostToDevice);
#else
    hipMemcpy(CudaData->d_Lint, CudaData->h_Lint, n_Total * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_Lxc, CudaData->h_Lxc, TotalEdge * sizeof(float), hipMemcpyHostToDevice);
#endif

#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_H2D2);
#endif
}

void CIRCUIT::MemoryCopy_D2H(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    //hipMemcpy(CudaData->h_Lcx, CudaData->d_Lcx, TotalEdge * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(CudaData->h_DecodedData, CudaData->d_DecodedData, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DecodedData, CudaData->d_DecodedData, n_Total * sizeof(int), hipMemcpyDeviceToHost);
#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_D2H);
#endif
}
#ifdef DOUBLE
__global__ void UpdateLcx_GPU(int* d_LxcSize, double* d_Lxc, int* d_LxcBegin, int* d_LcxPosition, double* d_Lcx,int k_Total){
#else
__global__ void UpdateLcx_GPU(int* d_LxcSize, float* d_Lxc, int* d_LxcBegin, int* d_LcxPosition, float* d_Lcx,int k_Total){
#endif
    int total_task = gridDim.x * blockDim.x;
    int task_sn = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=task_sn; i<k_Total; i+=total_task){
        for(int j=0; j<d_LxcSize[i]; j++){
#ifdef DOUBLE
            double sgn=1;
            double minLxc=1000;
#else
            float sgn=1;
            float minLxc=1000;
#endif
            for(int jj=0; jj<d_LxcSize[i]; jj++){
                if(j != jj){
                    if(d_Lxc[d_LxcBegin[i]+jj] > 0)
                        sgn = sgn*1;
                    else
                        sgn = sgn*(-1);
                    minLxc = min(minLxc, fabs(d_Lxc[d_LxcBegin[i]+jj]));
                }
            }
            d_Lcx[d_LcxPosition[d_LxcBegin[i]+j]] = sgn * minLxc;
        }
    }
}
#ifdef DOUBLE
__global__ void UpdateLxc_GPU(int* d_LcxSize, double* d_Lcx, int* d_LcxBegin, int* d_LxcPosition, double* d_Lxc, double* d_Lint, int n_Total){
#else
__global__ void UpdateLxc_GPU(int* d_LcxSize, float* d_Lcx, int* d_LcxBegin, int* d_LxcPosition, float* d_Lxc, float* d_Lint, int n_Total){
#endif
    int total_task = gridDim.x * blockDim.x;
    int task_sn = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=task_sn; i<n_Total; i+=total_task){
#ifdef DOUBLE
        double sumLcx=0;
#else
        float sumLcx=0;
#endif
        for(int j=0; j<d_LcxSize[i]; j++){
            sumLcx = sumLcx + d_Lcx[d_LcxBegin[i]+j];
        }
        for(int j=0; j<d_LcxSize[i]; j++){
             d_Lxc[d_LxcPosition[d_LcxBegin[i]+j]] = d_Lint[i] + (sumLcx - d_Lcx[d_LcxBegin[i]+j]);
        }
    }
}
#ifdef DOUBLE
__global__ void Calculate_Posterior_GPU(int* d_LcxSize, double* d_Lcx, int* d_LcxBegin, double* d_Lint, int* d_DecodedData,int n_Total){
#else
__global__ void Calculate_Posterior_GPU(int* d_LcxSize, float* d_Lcx, int* d_LcxBegin, float* d_Lint, int* d_DecodedData,int n_Total){
#endif
    int total_task = gridDim.x * blockDim.x;
    int task_sn = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=task_sn; i<n_Total; i+=total_task){
#ifdef DOUBLE
        double sumLcx=0;
#else
        float sumLcx=0;
#endif
        for(int j=0; j<d_LcxSize[i]; j++)
            sumLcx = sumLcx + d_Lcx[d_LcxBegin[i]+j];
        if(d_Lint[i] + sumLcx >= 0)
            d_DecodedData[i] = 0;
        else
            d_DecodedData[i] = 1;
    }
}

void CIRCUIT::Debug(){
    /*for(int i=0; i<m; i++){
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++){
            if(CheckNode[i]->Lxc[j] != CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j])
                cout<<CheckNode[i]->Lxc[j]<<"   "<<CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j]<<endl;
        }
    }
    for(int i=0; i<n; i++){
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){
            if(BitNode[i]->Lcx[j] != CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j])
                cout<<BitNode[i]->Lcx[j]<<" "<<CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j]<<endl;
        }
    }*/
    //cout<<"stop"<<endl;
    //getchar();
    for(int i=0; i<n; i++){
        if(DecodedData[i] != CudaData->h_DecodedData[i])
            cout<<DecodedData[i]<<" "<<CudaData->h_DecodedData[i]<<endl;
    }
}
