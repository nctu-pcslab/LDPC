#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <cstdio>
#include "circuit.h"
#include "device.h"
using namespace std;
//extern CIRCUIT Circuit;

void CIRCUIT::CUDA_MemoryAllocate(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    CudaData = new CUDA_DATA;
    CudaData->h_DecodedData = new int[n_Total];       //redundant, for CPU debug
    hipMalloc((void**)&CudaData->d_DecodedData, n_Total * sizeof(int));
    ////// BitNode on host //////
    CudaData->h_LcxSize = new int [n_Total];              //redundant?
    CudaData->h_LcxBegin = new int [n_Total];        
#ifdef DOUBLE
    CudaData->h_Lcx = new double [TotalEdge];
    CudaData->h_Lint = new double [TotalEdge];      //v2
#else
    CudaData->h_Lcx = new float [TotalEdge];
    CudaData->h_Lint = new float [TotalEdge];
#endif
    CudaData->h_NextLcxIndex = new int [TotalEdge];  //v2
    CudaData->h_LxcPosition = new int [TotalEdge];
    ////// CheckNode on host //////
    CudaData->h_LxcSize = new int [k_Total];              //redundant?
#ifdef DOUBLE
    CudaData->h_Lxc = new double [TotalEdge];
#else
    CudaData->h_Lxc = new float [TotalEdge];
#endif
    CudaData->h_LxcBegin = new int [k_Total];
    CudaData->h_NextLxcIndex = new int [TotalEdge];  //v2
    CudaData->h_LcxPosition = new int [TotalEdge];
    ////// BitNode on device //////
    hipMalloc((void**)&CudaData->d_LcxSize, n_Total * sizeof(int));  //redundant?
    hipMalloc((void**)&CudaData->d_LcxBegin, n_Total * sizeof(int)); //redundant?
#ifdef DOUBLE
    hipMalloc((void**)&CudaData->d_Lcx, TotalEdge * sizeof(double));
    hipMalloc((void**)&CudaData->d_Lint, TotalEdge * sizeof(double));
#else
    hipMalloc((void**)&CudaData->d_Lcx, TotalEdge * sizeof(float));
    hipMalloc((void**)&CudaData->d_Lint, TotalEdge * sizeof(float));
#endif
    hipMalloc((void**)&CudaData->d_NextLcxIndex, TotalEdge * sizeof(int));
    hipMalloc((void**)&CudaData->d_LxcPosition, TotalEdge * sizeof(int));
    ////// CheckNode on device //////
    hipMalloc((void**)&CudaData->d_LxcSize, k_Total * sizeof(int));  //redundant?
#ifdef DOUBLE
    hipMalloc((void**)&CudaData->d_Lxc, TotalEdge * sizeof(double));
#else
    hipMalloc((void**)&CudaData->d_Lxc, TotalEdge * sizeof(float));
#endif
    hipMalloc((void**)&CudaData->d_LxcBegin, k_Total * sizeof(int)); //redundant?
    hipMalloc((void**)&CudaData->d_NextLxcIndex, TotalEdge * sizeof(int));
    hipMalloc((void**)&CudaData->d_LcxPosition, TotalEdge * sizeof(int));
#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_MemoryAllocate);
#endif
}

void CIRCUIT::CUDA_CreateDataArray(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    int Begin=0;
    int Index=0;    //v2
    ///// BitNode /////
    for(unsigned i=0; i<n_Total; i++){
        CudaData->h_LcxSize[i] = BitNode[i]->Lcx.size();
        CudaData->h_LcxBegin[i] = Begin;
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){   //v2
            CudaData->h_NextLcxIndex[Index] = Index+1;
            Index++;
        }
        CudaData->h_NextLcxIndex[Index-1] = Begin;
        Begin=Begin+BitNode[i]->Lcx.size();
    }
    ///// CheckNode //////
    Begin=0;
    Index=0;    //v2
    for(unsigned i=0; i<k_Total; i++){
        CudaData->h_LxcSize[i] = CheckNode[i]->Lxc.size();
        CudaData->h_LxcBegin[i] = Begin;
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++){
            CudaData->h_LcxPosition[Begin+j] = CudaData->h_LcxBegin[CheckNode[i]->BitNode_H[j]->ID] + CheckNode[i]->Lcx_position[j];
            CudaData->h_NextLxcIndex[Index] = Index + 1;
            Index++;
        }
        CudaData->h_NextLxcIndex[Index-1] = Begin;
        Begin=Begin+CheckNode[i]->Lxc.size();
    }
    ///// BitNode /////
    for(unsigned i=0; i<n_Total; i++){
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++)
            CudaData->h_LxcPosition[CudaData->h_LcxBegin[i]+j] = CudaData->h_LxcBegin[BitNode[i]->CheckNode_H[j]->ID] + BitNode[i]->Lxc_position[j]; 
    }
    ///// Debug /////
    //Index = 0;
    /*for(unsigned i=0; i<n; i++){
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){   //v2
            cout<<CudaData->h_NextLcxIndex[Index]<<" ";
            Index++;
        }
        cout<<endl;
    }*/
    /*for(unsigned i=0; i<m; i++){
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++){
            cout<<CudaData->h_NextLxcIndex[Index]<<" ";
            Index++;
        }
        cout<<endl;
    }
    cout<<"TotalEdge = "<<TotalEdge<<endl;
    char a;
    cin>>a;*/
    //////// UpdateLcx_GPU /////////////
    CudaData->h_BlockBegin = new int[k_Total];
    //hipMalloc((void**)&CudaData->d_BlockBegin, (k_Total) * sizeof(int));
    int count=0;
    int count2=0;
    CudaData->TotalBlock = 1;
    CudaData->h_BlockBegin[0]=0;
    for(unsigned i=0; i<k_Total; i++){
        count=count+CheckNode[i]->Lxc.size();
        count2=count2+CheckNode[i]->Lxc.size();
        if(count > UCX_THD){
            CudaData->h_BlockBegin[CudaData->TotalBlock] = count2 - CheckNode[i]->Lxc.size();
            count = CheckNode[i]->Lxc.size();
            CudaData->TotalBlock++;
        }
    }
    CudaData->h_BlockBegin[CudaData->TotalBlock] = count2;
    hipMalloc((void**)&CudaData->d_BlockBegin, (CudaData->TotalBlock+1) * sizeof(int));
    hipMemcpy(CudaData->d_BlockBegin, CudaData->h_BlockBegin, (CudaData->TotalBlock+1) * sizeof(int), hipMemcpyHostToDevice);
    ///////// debug /////////
    /*for(unsigned i=0; i<CudaData->TotalBlock-1; i++)
        cout<<"Begin:"<<CudaData->h_BlockBegin[i]<<"  Size:"<<CudaData->h_BlockBegin[i+1]-CudaData->h_BlockBegin[i]<<endl;
    cout<<"Begin:"<<CudaData->h_BlockBegin[CudaData->TotalBlock-1]<<"  Size:"<<CudaData->h_BlockBegin[CudaData->TotalBlock]-CudaData->h_BlockBegin[CudaData->TotalBlock-1]<<endl;
    for(unsigned i=0; i<CudaData->TotalBlock+1; i++)
        cout<<CudaData->h_BlockBegin[i]<<endl;
    getchar();*/
    //////// UpdateLxc_GPU /////////////
    CudaData->h_BlockBeginUXC = new int[n_Total];
    count=0;
    count2=0;
    CudaData->TotalBlockUXC = 1;
    CudaData->h_BlockBeginUXC[0]=0;
    for(unsigned i=0; i<n_Total; i++){
        count=count+BitNode[i]->Lcx.size();
        count2=count2+BitNode[i]->Lcx.size();
        if(count > UXC_THD){
            CudaData->h_BlockBeginUXC[CudaData->TotalBlockUXC] = count2 - BitNode[i]->Lcx.size();
            count = BitNode[i]->Lcx.size();
            CudaData->TotalBlockUXC++;
        }
    }
    CudaData->h_BlockBeginUXC[CudaData->TotalBlockUXC] = count2;
    hipMalloc((void**)&CudaData->d_BlockBeginUXC, (CudaData->TotalBlockUXC+1) * sizeof(int));
    hipMemcpy(CudaData->d_BlockBeginUXC, CudaData->h_BlockBeginUXC, (CudaData->TotalBlockUXC+1) * sizeof(int), hipMemcpyHostToDevice);
    ///////// debug /////////
    /*for(unsigned i=0; i<CudaData->TotalBlockUXC-1; i++)
        cout<<"Begin:"<<CudaData->h_BlockBeginUXC[i]<<"  Size:"<<CudaData->h_BlockBeginUXC[i+1]-CudaData->h_BlockBeginUXC[i]<<endl;
    cout<<"Begin:"<<CudaData->h_BlockBeginUXC[CudaData->TotalBlockUXC-1]<<"  Size:"<<CudaData->h_BlockBeginUXC[CudaData->TotalBlockUXC]-CudaData->h_BlockBeginUXC[CudaData->TotalBlockUXC-1]<<endl;
    for(unsigned i=0; i<CudaData->TotalBlockUXC+1; i++)
        cout<<CudaData->h_BlockBeginUXC[i]<<endl;
    getchar();*/
#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_CreateDataArray);
#endif
}

void CIRCUIT::CUDA_CreateDataArray2(){
#ifdef PROFILE
    Timer.TimerStart();
#endif

    for(unsigned i=0; i<n_Total; i++){
        //CudaData->h_Lint[i] = BitNode[i]->Lint;
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){
            //CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j] = BitNode[i]->Lcx[j];
            CudaData->h_Lint[CudaData->h_LcxBegin[i]+j] = BitNode[i]->Lint;
        }
    }
    for(unsigned i=0; i<k_Total; i++)
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++)
            CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j] = CheckNode[i]->Lxc[j];

#ifdef PROFILE
    Timer.TimerFinish(Timer.tCUDA_CreateDataArray2);
#endif
}

void CIRCUIT::UpdateLcx_CPU(){
    for(int i=0; i<TotalEdge; i++){
        double sgn=1;
        double minLxc=1000;
        int Index = CudaData->h_NextLxcIndex[i];
        while(Index != i){
            if(CudaData->h_Lxc[Index] > 0)
                sgn = sgn*1;
            else
                sgn = sgn*(-1);
            minLxc = min(minLxc, fabs(CudaData->h_Lxc[Index]));
            Index = CudaData->h_NextLxcIndex[Index];
        }
        CudaData->h_Lcx[CudaData->h_LcxPosition[i]] = sgn * minLxc;        
    }
}

void CIRCUIT::UpdateLxc_CPU(){
    for(int i=0; i<TotalEdge; i++){
        double sumLcx=0;
        int Index = CudaData->h_NextLcxIndex[i];
        while(Index != i){
            sumLcx = sumLcx + CudaData->h_Lcx[Index];
            Index = CudaData->h_NextLcxIndex[Index];
        }
        CudaData->h_Lxc[CudaData->h_LxcPosition[i]] = CudaData->h_Lint[i] + sumLcx;
    }
}

void CIRCUIT::Calculate_Posterior_CPU(){
    for(int i=0; i<n_Total; i++){
        double sumLcx=0;
        for(int j=0; j<CudaData->h_LcxSize[i]; j++)
            sumLcx = sumLcx + CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j];
        if(CudaData->h_Lint[CudaData->h_LcxBegin[i]] + sumLcx >= 0)
            CudaData->h_DecodedData[i] = 0;
        else
            CudaData->h_DecodedData[i] = 1;
    } 
}

void CIRCUIT::MemoryCopy_H2D(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    ////// BitNode on device //////
    hipMemcpy(CudaData->d_LcxSize, CudaData->h_LcxSize, n_Total * sizeof(int), hipMemcpyHostToDevice);//redundant
    hipMemcpy(CudaData->d_LcxBegin, CudaData->h_LcxBegin, n_Total * sizeof(int), hipMemcpyHostToDevice);//redundant
    hipMemcpy(CudaData->d_NextLcxIndex, CudaData->h_NextLcxIndex, TotalEdge * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LxcPosition, CudaData->h_LxcPosition, TotalEdge * sizeof(int), hipMemcpyHostToDevice);
    ////// CheckNode on device //////
    hipMemcpy(CudaData->d_LxcSize, CudaData->h_LxcSize, k_Total * sizeof(int), hipMemcpyHostToDevice);//redundant
    hipMemcpy(CudaData->d_LxcBegin, CudaData->h_LxcBegin, k_Total * sizeof(int), hipMemcpyHostToDevice);//redundant
    hipMemcpy(CudaData->d_NextLxcIndex, CudaData->h_NextLxcIndex, TotalEdge * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_LcxPosition, CudaData->h_LcxPosition, TotalEdge * sizeof(int), hipMemcpyHostToDevice);

#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_H2D);
#endif
}

void CIRCUIT::MemoryCopy_H2D2(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
#ifdef DOUBLE
    ////// BitNode on device //////
    //hipMemcpy(CudaData->d_Lcx, CudaData->h_Lcx, TotalEdge * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_Lint, CudaData->h_Lint, TotalEdge * sizeof(double), hipMemcpyHostToDevice);
    ////// CheckNode on device //////
    hipMemcpy(CudaData->d_Lxc, CudaData->h_Lxc, TotalEdge * sizeof(double), hipMemcpyHostToDevice);
#else
    hipMemcpy(CudaData->d_Lint, CudaData->h_Lint, TotalEdge * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(CudaData->d_Lxc, CudaData->h_Lxc, TotalEdge * sizeof(float), hipMemcpyHostToDevice);
#endif
#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_H2D2);
#endif
}

void CIRCUIT::MemoryCopy_D2H(){
#ifdef PROFILE
    Timer.TimerStart();
#endif
    //hipMemcpy(CudaData->h_Lcx, CudaData->d_Lcx, TotalEdge * sizeof(double), hipMemcpyDeviceToHost);
    //hipMemcpy(CudaData->h_DecodedData, CudaData->d_DecodedData, n * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(DecodedData, CudaData->d_DecodedData, n_Total * sizeof(int), hipMemcpyDeviceToHost);
#ifdef PROFILE
    Timer.TimerFinish(Timer.tMemoryCopy_D2H);
#endif
}
#ifdef DOUBLE
__global__ void UpdateLcx_GPU(double* d_Lxc, int* d_NextLxcIndex, int* d_LcxPosition, double* d_Lcx, int TotalEdge, int* d_BlockBegin, int TotalBlock){
#else
__global__ void UpdateLcx_GPU(float* d_Lxc, int* d_NextLxcIndex, int* d_LcxPosition, float* d_Lcx, int TotalEdge, int* d_BlockBegin, int TotalBlock){
#endif
#ifdef DOUBLE
    __shared__ double s_Lxc[UCX_THD];
#else
    __shared__ float s_Lxc[UCX_THD];
#endif
    __shared__ int s_NextLxcIndex[UCX_THD];
    for(int i=blockIdx.x; i<TotalBlock; i+=gridDim.x){
        if(threadIdx.x < d_BlockBegin[i+1] - d_BlockBegin[i]){
            int task_sn = d_BlockBegin[i]+threadIdx.x;
            s_Lxc[threadIdx.x] = d_Lxc[task_sn];
            s_NextLxcIndex[threadIdx.x] = d_NextLxcIndex[task_sn] - (task_sn - threadIdx.x);
            __syncthreads();
            
#ifdef DOUBLE
            double sgn=1;
            double minLxc=1000;
#else
            float sgn=1;
            float minLxc=1000;
#endif
            int Index = s_NextLxcIndex[threadIdx.x];
            while(Index != threadIdx.x){
                if(s_Lxc[Index] < 0){
                    sgn = sgn*(-1);
                }
                minLxc = min(minLxc, fabs(s_Lxc[Index]));
                Index = s_NextLxcIndex[Index];
            }
            d_Lcx[d_LcxPosition[task_sn]] = sgn * minLxc;
        }
        //__syncthreads();
    }
    /*for(int i=blockIdx.x; i<TotalBlock; i+=gridDim.x){
        if(threadIdx.x < d_BlockBegin[i+1] - d_BlockBegin[i]){
            float sgn=1;
            float minLxc=1000;
            int task_sn = d_BlockBegin[i]+threadIdx.x;
            int Index = d_NextLxcIndex[task_sn];
            while(Index != task_sn){
                if(d_Lxc[Index] < 0)
                    sgn = sgn*(-1);
                minLxc = min(minLxc, fabs(d_Lxc[Index]));
                Index = d_NextLxcIndex[Index];        
            }
            d_Lcx[d_LcxPosition[task_sn]] = sgn * minLxc;
        }
    }*/
}
#ifdef DOUBLE
__global__ void UpdateLxc_GPU(double* d_Lcx, int* d_NextLcxIndex, int* d_LxcPosition, double* d_Lxc, double* d_Lint, int TotalEdge, int* d_BlockBegin, int TotalBlock){
#else
__global__ void UpdateLxc_GPU(float* d_Lcx, int* d_NextLcxIndex, int* d_LxcPosition, float* d_Lxc, float* d_Lint, int TotalEdge, int* d_BlockBegin, int TotalBlock){
#endif
/*    int total_task = gridDim.x * blockDim.x;
    int task_sn = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=task_sn; i<TotalEdge; i+=total_task){
#ifdef DOUBLE
        double sumLcx=0;
#else
        float sumLcx=0;
#endif
        int Index = d_NextLcxIndex[i];
        while(Index != i){
            sumLcx = sumLcx + d_Lcx[Index];
            Index = d_NextLcxIndex[Index];
        }
        d_Lxc[d_LxcPosition[i]] = d_Lint[i] + sumLcx;
    }*/

    __shared__ float s_Lcx[UXC_THD];
    __shared__ int s_NextLcxIndex[UXC_THD];

    /*for(int i=blockIdx.x; i<TotalBlock; i+=gridDim.x){
        if(threadIdx.x < d_BlockBegin[i+1] - d_BlockBegin[i]){
            int task_sn = d_BlockBegin[i]+threadIdx.x;
            float sumLcx=0;
            s_Lcx[threadIdx.x] = d_Lcx[task_sn];
            s_NextLcxIndex[threadIdx.x] = d_NextLcxIndex[task_sn];
            __syncthreads();
            
            int Index = s_NextLcxIndex[threadIdx.x];
            while(Index != task_sn){
                sumLcx = sumLcx + s_Lcx[Index - task_sn + threadIdx.x];
                Index = s_NextLcxIndex[Index - task_sn + threadIdx.x];
            }
            d_Lxc[d_LxcPosition[task_sn]] = d_Lint[task_sn] + sumLcx;
        }
        __syncthreads();
    }*/
    /*for(int i=blockIdx.x; i<TotalBlock; i+=gridDim.x){
        if(threadIdx.x < d_BlockBegin[i+1] - d_BlockBegin[i]){
            int task_sn = d_BlockBegin[i]+threadIdx.x;
            float sumLcx=0;
            s_Lcx[threadIdx.x] = d_Lcx[task_sn];
            s_NextLcxIndex[threadIdx.x] = d_NextLcxIndex[task_sn];
            __syncthreads();
            s_NextLcxIndex[threadIdx.x] = s_NextLcxIndex[threadIdx.x] - task_sn + threadIdx.x;
            __syncthreads();
            int Index = s_NextLcxIndex[threadIdx.x];
            //while(Index != task_sn){
            while(Index != threadIdx.x){
                sumLcx = sumLcx + s_Lcx[Index];
                Index = s_NextLcxIndex[Index];
            }
            d_Lxc[d_LxcPosition[task_sn]] = d_Lint[task_sn] + sumLcx;
        }
    }*/
    for(int i=blockIdx.x; i<TotalBlock; i+=gridDim.x){
       /* int task_sn = d_BlockBegin[i]+threadIdx.x; 
        s_Lcx[threadIdx.x] = d_Lcx[task_sn];
            s_NextLcxIndex[threadIdx.x] = d_NextLcxIndex[task_sn] - (task_sn - threadIdx.x);
            __syncthreads();*/
        if(threadIdx.x < d_BlockBegin[i+1] - d_BlockBegin[i]){
            int task_sn = d_BlockBegin[i]+threadIdx.x;
            s_Lcx[threadIdx.x] = d_Lcx[task_sn];
            s_NextLcxIndex[threadIdx.x] = d_NextLcxIndex[task_sn] - (task_sn - threadIdx.x);
            __syncthreads();
            
            float sumLcx=0;
            int Index = s_NextLcxIndex[threadIdx.x];
            while(Index != threadIdx.x){
                sumLcx = sumLcx + s_Lcx[Index];
                Index = s_NextLcxIndex[Index];
            }
            d_Lxc[d_LxcPosition[task_sn]] = d_Lint[task_sn] + sumLcx;
        }
    }
}
#ifdef DOUBLE
__global__ void Calculate_Posterior_GPU(int* d_LcxSize, double* d_Lcx, int* d_LcxBegin, double* d_Lint, int* d_DecodedData,int n_Total){
#else
__global__ void Calculate_Posterior_GPU(int* d_LcxSize, float* d_Lcx, int* d_LcxBegin, float* d_Lint, int* d_DecodedData,int n_Total){
#endif
    int total_task = gridDim.x * blockDim.x;
    int task_sn = blockIdx.x*blockDim.x + threadIdx.x;

    for(int i=task_sn; i<n_Total; i+=total_task){
#ifdef DOUBLE
        double sumLcx=0;
#else
        float sumLcx=0;
#endif
        for(int j=0; j<d_LcxSize[i]; j++)
            sumLcx = sumLcx + d_Lcx[d_LcxBegin[i]+j];
        if(d_Lint[d_LcxBegin[i]] + sumLcx >= 0)
            d_DecodedData[i] = 0;
        else
            d_DecodedData[i] = 1;
    }
}

void CIRCUIT::Debug(){
    /*for(int i=0; i<m; i++){
        for(unsigned j=0; j<CheckNode[i]->Lxc.size(); j++){
            if(CheckNode[i]->Lxc[j] != CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j])
                cout<<CheckNode[i]->Lxc[j]<<"   "<<CudaData->h_Lxc[CudaData->h_LxcBegin[i]+j]<<endl;
        }
    }
    for(int i=0; i<n; i++){
        for(unsigned j=0; j<BitNode[i]->Lcx.size(); j++){
            if(BitNode[i]->Lcx[j] != CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j])
                cout<<BitNode[i]->Lcx[j]<<" "<<CudaData->h_Lcx[CudaData->h_LcxBegin[i]+j]<<endl;
        }
    }*/
    //cout<<"stop"<<endl;
    //getchar();
    for(int i=0; i<n_Total; i++){
        if(DecodedData[i] != CudaData->h_DecodedData[i])
            cout<<DecodedData[i]<<" "<<CudaData->h_DecodedData[i]<<endl;
    }
}
